#include "hip/hip_runtime.h"
/*


   Copyright 2016  Amazon.com, Inc. or its affiliates. All Rights Reserved.

   Licensed under the Apache License, Version 2.0 (the "License"). You may not use this file except in compliance with the License. A copy of the License is located at

   http://aws.amazon.com/apache2.0/

   or in the "license" file accompanying this file. This file is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the License for the specific language governing permissions and limitations under the License.
 */

#include "GpuTypes.h"
#include "NNTypes.h"
#include <limits>

static __constant__ GpuData cData;

void SetKLossGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

void GetKLossGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));     
    RTERROR(status, "hipMemcpyFromSymbol: SetKernelsGpuData copy From cData failed");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawL1Error_kernel(uint32_t position, NNFloat* pDataWeight, NNFloat* pUnit, uint64_t stride, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = (NNFloat)1.0;
        if (pDataWeight != NULL)
        {
            uint64_t dpos       = (pos / stride) + position;
            dpos                = cData._bShuffleIndices ? cData._pShuffleIndex[dpos] : dpos;
            w                  *= pDataWeight[dpos];
        }

        NNFloat a               = pUnit[pos];
        error                   = w * fabsf(a);     
    }
    
    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        uint64_t offset         = pos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * (fabsf(a - (NNFloat)1.0) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * fabsf(a - (NNFloat)1.0);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}



NNFloat kCalculateSparseL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateSparseOnlyNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroL1Error_kernel");    
    }
    else
    {
        uint64_t size               = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks             = CalculateBlocks(size);    
        kCalculateSparseRawL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL1Error_kernel");
        blocks                      = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseNonZeroL1Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * (fabsf(a - (NNFloat)1.0) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * fabsf(a - (NNFloat)1.0);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateIndexedSparseL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateIndexedSparseOnlyNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseOnlyNonZeroL1Error_kernel");    
    }
    else
    {
        uint64_t size               = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks             = CalculateBlocks(size);    
        kCalculateSparseRawL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL1Error_kernel");
        blocks                      = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseNonZeroL1Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateSparseAnalogL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateSparseAnalogOnlyNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogOnlyNonZeroL1Error_kernel");   
    }
    else
    {
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL1Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseAnalogNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogNonZeroL1Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * fabsf(a - t);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL1Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * (fabsf(a - t) - fabsf(a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateIndexedSparseAnalogL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateIndexedSparseAnalogOnlyNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogOnlyNonZeroL1Error_kernel");   
    }
    else
    {
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL1Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseAnalogNonZeroL1Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateIndexedSparseAnalogNonZeroL1Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawL2Error_kernel(uint32_t position, NNFloat* pDataWeight, NNFloat* pUnit, uint32_t stride, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = (NNFloat)0.5;
        if (pDataWeight != NULL)
        {
            uint64_t dpos       = (pos / stride) + position;
            dpos                = cData._bShuffleIndices ? cData._pShuffleIndex[dpos] : dpos;
            w                  *= pDataWeight[dpos];
        }
        NNFloat a               = pUnit[pos];
        error                   = w * a * a;     
    }
    
    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * ((a - (NNFloat)1.0) * (a - (NNFloat)1.0));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * ((a - (NNFloat)1.0) * (a - (NNFloat)1.0) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


NNFloat kCalculateSparseL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateSparseOnlyNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroL2Error_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseNonZeroL2Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateSparseAnalogL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseAnalogOnlyNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogOnlyNonZeroL2Error_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseAnalogNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogNonZeroL2Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * ((a - (NNFloat)1.0) * (a - (NNFloat)1.0));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * ((a - (NNFloat)1.0) * (a - (NNFloat)1.0) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


NNFloat kCalculateIndexedSparseL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateIndexedSparseOnlyNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseOnlyNonZeroL2Error_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseNonZeroL2Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * ((a - t) * (a - t));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2Error_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * ((a - t) * (a - t) - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateIndexedSparseAnalogL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseAnalogOnlyNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateIndexedSparseAnalogOnlyNonZeroL2Error_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2Error_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseAnalogNonZeroL2Error_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateIndexedSparseAnalogNonZeroL2Error_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}


__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawL2HingeError_kernel(uint32_t position, NNFloat* pDataWeight, NNFloat* pUnit, uint32_t stride, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = (NNFloat)0.5;
        if (pDataWeight != NULL)
        {
            uint64_t dpos       = (pos / stride) + position;
            dpos                = cData._bShuffleIndices ? cData._pShuffleIndex[dpos] : dpos;
            w                  *= pDataWeight[dpos];
        }
        NNFloat a               = max((NNFloat)0.0, pUnit[pos]);
        error                   = w * a * a;
    }
    
    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat diff        = min((NNFloat)0.0, a - (NNFloat)1.0);
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat diff        = min((NNFloat)0.0, a - (NNFloat)1.0);
            a                   = max((NNFloat)0.0, a);
            error              += w * (diff * diff - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateSparseL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateSparseOnlyNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroL2HingeError_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2HingeError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseNonZeroL2HingeError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            NNFloat diff        = a - fabsf(t);
            diff                = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            NNFloat diff        = a - fabsf(t);
            a                   = max((NNFloat)0.0, a);
            diff                = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);          
            error              += w * (diff * diff - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            NNFloat diff        = a - t;
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff);             
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            NNFloat diff        = a - t;
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff);   
            a                   = max((NNFloat)0.0, a);  
            error              += w * (diff * diff - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            NNFloat diff        = a - fabsf((NNFloat)t);
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);           
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            NNFloat diff        = a - fabsf(t);
            a                   = max((NNFloat)0.0, a);
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);       
            error              += w * (diff * diff - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateSparseAnalogL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseAnalogOnlyNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogOnlyNonZeroL2HingeError_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2HingeError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseAnalogNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateSparseAnalogNonZeroL2HingeError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat diff        = min((NNFloat)0.0, pUnit[pos2] - (NNFloat)1.0);
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat diff        = min((NNFloat)0.0, a - (NNFloat)1.0);
            a                   = max((NNFloat)0.0, a);
            error              += w * (diff * diff - a * a); 
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


NNFloat kCalculateIndexedSparseL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);    
        kCalculateIndexedSparseOnlyNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseOnlyNonZeroL2HingeError_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2HingeError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseNonZeroL2HingeError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            NNFloat diff        = a - fabsf(t);
            diff                = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);         
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            NNFloat diff        = a - fabsf(t);
            a                   = max((NNFloat)0.0, a);
            diff                = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);          
            error              += w * (diff * diff - a * a);               
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            NNFloat diff        = a - t;
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff); 
            error              += w * diff * diff;   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            NNFloat diff        = a - t;
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff);    
            a                   = max((NNFloat)0.0, a);  
            error              += w * (diff * diff - a * a);   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogOnlyNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            NNFloat diff        = a - fabsf(t);
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);      
            error              += w * diff * diff;  
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogNonZeroL2HingeError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            NNFloat diff        = a - fabsf(t);
            a                   = max((NNFloat)0.0, a);
            diff                = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);          
            error              += w * (diff * diff - a * a);  
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateIndexedSparseAnalogL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseAnalogOnlyNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateIndexedSparseAnalogOnlyNonZeroL2HingeError_kernel");    
    }
    else
    {
        uint64_t size           = batch * stride;
        uint32_t blocks         = CalculateBlocks(size);    
        kCalculateSparseRawL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawL2HingeError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseAnalogNonZeroL2HingeError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
        LAUNCHERROR("kCalculateIndexedSparseAnalogNonZeroL2HingeError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawCrossEntropyError_kernel(uint32_t position, NNFloat* pDataWeight, NNFloat* pUnit, uint32_t stride, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = (NNFloat)1.0;
        if (pDataWeight != NULL)
        {
            uint64_t dpos       = (pos / stride) + position;
            dpos                = cData._bShuffleIndices ? cData._pShuffleIndex[dpos] : dpos;
            w                  *= pDataWeight[dpos];
        }
        NNFloat a               = pUnit[pos];
        error                   = -w * log(max(MIN_ERROR, (NNFloat)1.0 - a));     
    }

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseOnlyNonZeroCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += -w * log(max(MIN_ERROR, a));   
            pos1               += cData._warpSize;
        }
    }  

/* LOOPY
            while (pos1 < end)
            {
                uint64_t pos2       = offset + pSparseIndex[pos1];
                NNFloat a           = pUnit[pos2];
                error              += -t * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a));   
                pos1               += cData._warpSize;
            }
*/

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * (-log(max(MIN_ERROR, a)) + log(max(MIN_ERROR, (NNFloat)1.0 - a)));   
            pos1               += cData._warpSize;
        }
/* LOOPY
            while (pos1 < end)
            {
                uint64_t pos2       = offset + pSparseIndex[pos1];
                NNFloat a           = pUnit[pos2];
                error              += -t * log(max(MIN_ERROR, a)) + t * log(max(MIN_ERROR, (NNFloat)1.0 - a)); // -t * log(a) - (1.0 - t) * log(1.0 - a) + log(1.0 - a)  
                pos1               += cData._warpSize;
            }
*/
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateSparseCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseOnlyNonZeroCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroCrossEntropyError_kernel");    
    }
    else
    {    
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);
        kCalculateSparseRawCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawCrossEntropyError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseNonZeroCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseNonZeroCrossEntropyError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);

    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseOnlyNonZeroCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += -w * log(max(MIN_ERROR, a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += w * (-log(max(MIN_ERROR, a)) + log(max(MIN_ERROR, (NNFloat)1.0 - a)));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateIndexedSparseCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseOnlyNonZeroCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroCrossEntropyError_kernel");    
    }
    else
    {    
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);
        kCalculateSparseRawCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawCrossEntropyError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseNonZeroCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseNonZeroCrossEntropyError_kernel");
    }
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);

    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos];
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight == NULL) ? (NNFloat)1.0 / (NNFloat)(end - pos1) : pDataWeight[dpos];
        pos1                   += threadIdx.x & cData._warpMask;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += -w * log(max(MIN_ERROR, a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateSparseMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseMultinomialCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kCalculateSparseMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);

    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos];
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight == NULL) ? (NNFloat)1.0 / (NNFloat)(end - pos1) : pDataWeight[dpos];
        pos1                   += threadIdx.x & cData._warpMask;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            error              += -w * log(max(MIN_ERROR, a));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateIndexedSparseMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseMultinomialCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kCalculateIndexedSparseMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);

    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * (-t * log(max(MIN_ERROR, a)));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * (-t * log(max(MIN_ERROR, a)));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * (-t * log(max(MIN_ERROR, a)));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateSparseAnalogMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseAnalogMultinomialCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kCalculateSparseAnalogMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];
            error              += w * (-t * log(max(MIN_ERROR, a)));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            error              += w * (-t * log(max(MIN_ERROR, a)));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{

    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = (NNFloat)pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            error              += w * (-t * log(max(MIN_ERROR, a)));   
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}


template<typename T>
NNFloat kCalculateIndexedSparseAnalogMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseAnalogMultinomialCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kCalculateIndexedSparseAnalogMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawScaledMarginalCrossEntropyError_kernel(uint32_t position, NNFloat* pDataWeight, NNFloat* pUnit, uint32_t stride, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = cData._SMCE_zeroScale;
        if (pDataWeight != NULL)
        {
            uint64_t dpos       = pos / stride;
            dpos                = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
            w                  *= pDataWeight[dpos];
        }
        NNFloat a               = pUnit[pos];
        if (a > cData._SMCE_zeroTarget)
            error               = -w * log(max(MIN_ERROR, (NNFloat)1.0 - a));     
    }
    
    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            if (a < cData._SMCE_oneTarget)
                error          += -w * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
/* LOOPY
        }
        else
        {
            while (pos1 < end)
            {
                uint64_t pos2       = offset + pSparseIndex[pos1];
                NNFloat a           = pUnit[pos2];
                if (a < cData._SMCE_oneTarget)
                   error           += cData._SMCE_oneScale * (-t * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));   
                pos1               += cData._warpSize;
            }
        }
*/
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            if (a > cData._SMCE_zeroTarget)
            {
                error          += w * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
            }
            if (a < cData._SMCE_oneTarget)
            {
                error          += -w * cData._SMCE_oneScale * log(max(MIN_ERROR, a));
            }
            pos1               += cData._warpSize;
        }


/* LOOPY
        }
        else
        {
            while (pos1 < end)
            {
                uint64_t pos2       = offset + pSparseIndex[pos1];
                NNFloat a           = pUnit[pos2];
                if (a > cData._SMCE_zeroTarget)
                {
                    error          += cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
                }
                if (a < cData._SMCE_oneTarget)
                {
                    error          += cData._SMCE_oneScale * (-t * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));
                }
                pos1               += cData._warpSize;
            }
        }
*/
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateSparseScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel");   
    }
    else
    {
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);
        kCalculateSparseRawScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawScaledMarginalCrossEntropyError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateSparseNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseNonZeroScaledMarginalCrossEntropyError_kernel");
    }    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            if (a < cData._SMCE_oneTarget)
                error          += -w * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            if (a > cData._SMCE_zeroTarget)
            {
                error          += w * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
            }
            if (a < cData._SMCE_oneTarget)
            {
                error          += -w * cData._SMCE_oneScale * log(max(MIN_ERROR, a));
            }
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateIndexedSparseScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    if (bSparseIgnoreZero)
    {
        uint32_t blocks         = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateSparseOnlyNonZeroScaledMarginalCrossEntropyError_kernel");   
    }
    else
    {
        uint64_t size           = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks         = CalculateBlocks(size);
        kCalculateSparseRawScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, pDataWeight, pUnit, stride, size);
        LAUNCHERROR("kCalculateSparseRawScaledMarginalCrossEntropyError_kernel");
        blocks                  = CalculateBlocks(batch * getGpu()._warpSize);
        kCalculateIndexedSparseNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
        LAUNCHERROR("kCalculateIndexedSparseNonZeroScaledMarginalCrossEntropyError_kernel");
    }    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseRawDataScaledMarginalCrossEntropyError_kernel(NNFloat* pUnit, uint64_t size)
{
    uint64_t pos                = blockDim.x * blockIdx.x + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
          NNFloat a               = pUnit[pos];
          if (a > cData._SMCE_zeroTarget)
          {
              error               = -cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
          }
    }

    REDUCEERROR(error)
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseNonZeroDataScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, T* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
              uint64_t pos2       = offset + pSparseIndex[pos1];
              NNFloat a           = pUnit[pos2];
              T t                 = pSparseData[pos1];

              if (a > cData._SMCE_zeroTarget)
              {
                  error          += cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
              }

              if (a < cData._SMCE_oneTarget)
              {
                  error          += -cData._SMCE_oneScale * t * log(max(MIN_ERROR, a));
              }
              pos1               += cData._warpSize;
        }
    }

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateSparseDataScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));

    if (!bSparseIgnoreZero)
    {
        uint64_t size               = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks             = CalculateBlocks(size);
        kCalculateSparseRawDataScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, size);
        LAUNCHERROR("kCalculateSparseRawDataScaledMarginalCrossEntropyError_kernel");
    }
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseNonZeroDataScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pSparseData);
    LAUNCHERROR("kCalculateSparseNonZeroDataScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}



template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseNonZeroDataScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, T* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
              uint64_t pos2       = offset + pSparseIndex[pos1];
              NNFloat a           = pUnit[pos2];
              T t                 = pSparseData[pos1];

              if (a > cData._SMCE_zeroTarget)
              {
                  error          += cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a));
              }

              if (a < cData._SMCE_oneTarget)
              {
                  error          += -cData._SMCE_oneScale * t * log(max(MIN_ERROR, a));
              }
              pos1               += cData._warpSize;
        }
    }

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateIndexedSparseDataScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, T* pSparseData, bool bSparseIgnoreZero)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));

    if (!bSparseIgnoreZero)
    {
        uint64_t size               = (uint64_t)batch * (uint64_t)stride;
        uint32_t blocks             = CalculateBlocks(size);
        kCalculateSparseRawDataScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, size);
        LAUNCHERROR("kCalculateSparseRawDataScaledMarginalCrossEntropyError_kernel");
    }
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseNonZeroDataScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pSparseData);
    LAUNCHERROR("kCalculateIndexedSparseNonZeroDataScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos];
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight == NULL) ? (NNFloat)1.0 / (NNFloat)(end - pos1) : pDataWeight[dpos]);
        pos1                   += threadIdx.x & cData._warpMask;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2]; 
            if (a < cData._SMCE_oneTarget)
                error          += -w * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateSparseMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kCalculateSparseMultinomialScaledMarginalCrossEntropyError_kernel");    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos];
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight == NULL) ? (NNFloat)1.0 / (NNFloat)(end - pos1) : pDataWeight[dpos]);
        pos1                   += threadIdx.x & cData._warpMask;
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2]; 
            if (a < cData._SMCE_oneTarget)
                error          += -w * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

NNFloat kCalculateIndexedSparseMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseNonZeroScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kCalculateIndexedSparseMultinomialScaledMarginalCrossEntropyError_kernel");    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}




template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];  
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos;
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel");    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}


template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            T t                 = pSparseData[pos1];  
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = pSparseData[pos1] * (NNFloat)(1.0 / 256.0);
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat *pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData)
{
    uint64_t pos                = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    NNFloat error               = (NNFloat)0.0;
    if (pos < batch)
    {
        uint32_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + pos] : position + pos];
        uint64_t pos1           = pSparseStart[dpos] + (threadIdx.x & cData._warpMask);
        uint64_t end            = pSparseEnd[dpos];
        NNFloat w               = cData._SMCE_oneScale * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        uint64_t offset         = pos * stride;
        while (pos1 < end)
        {
            uint64_t pos2       = offset + pSparseIndex[pos1];
            NNFloat a           = pUnit[pos2];
            NNFloat t           = pSparseData[pos1] * (NNFloat)(1.0 / 128.0);
            if (a < cData._SMCE_oneTarget)
                error          += -w * t * log(max(MIN_ERROR, a));
            pos1               += cData._warpSize;
        }
    }  

    REDUCEERROR(error)
}

template<typename T>
NNFloat kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t *pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError_kernel");    
    getGpu()._pbAccumulator->Download(); 
    //printf("Error is %f\n",  (double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;    
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;        
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;            
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateL1Error_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateL1Error_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL1Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0; 
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * fabsf(a - t);        
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedL1Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedL1Error_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateL1Error_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (a - t) * (a - t);         
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (a - t) * (a - t);         

    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (a - t) * (a - t);         

    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateL2Error_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateL2Error_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (a - t) * (a - t);         
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (a - t) * (a - t);         

    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2Error_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (a - t) * (a - t);         

    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedL2Error(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedL2Error_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedL2Error_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);        
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        NNFloat diff            = a - fabsf(t);
        diff                    = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);
        error                  += w * diff * diff;   
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        NNFloat diff            = a - t;
        diff                    = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff);      
        error                   = w * diff * diff;         
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        NNFloat diff            = a - fabsf(t);
        diff                    = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);
        error                  += w * diff * diff;       
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateL2HingeError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateL2HingeError_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        NNFloat diff            = a - fabsf(t);
        diff                    = (t > (T)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);
        error                  += w * diff * diff;           
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        NNFloat diff            = a - t;
        diff                    = (t > (NNFloat)0.0) ? min((NNFloat)0.0f, diff) : max((NNFloat)0.0, diff);      
        error                   = w * diff * diff;        
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedL2HingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (NNFloat)0.5 * ((pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0);
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        NNFloat diff            = a - fabsf(t);
        diff                    = (t > (NNFloat)0.0) ? min((NNFloat)0.0f , diff) : max((NNFloat)0.0, diff);
        error                  += w * diff * diff;        
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedL2HingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedL2HingeError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedL2HingeError_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    
    while (pos < stride)
    {
        NNFloat t               = pData[pos];
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;
        //printf("HL %d %f %f %f\n", blockIdx.x, t, y, loss);
    }
    
    REDUCEERROR(loss)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{ 
    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    
    while (pos < stride)
    {
        NNFloat t               = pData[pos] * (NNFloat)(1.0 / 128.0);
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;    
    }
    
    REDUCEERROR(loss)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{

    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    
    while (pos < stride)
    {
        NNFloat t               = (NNFloat)pData[pos] * (NNFloat)(1.0 / 256.0);
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;    
    }
    
    REDUCEERROR(loss)
}

template<typename T> NNFloat kCalculateHingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    unsigned long threads = max(32, min(stride, 128));
    kCalculateHingeError_kernel<<<batch, threads>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateHingeError_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    
    while (pos < stride)
    {
        NNFloat t               = pData[pos];
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;    
    }
    
    REDUCEERROR(loss)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{ 
    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    while (pos < stride)
    {
        NNFloat t               = pData[pos] * (NNFloat)(1.0 / 256.0);
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;    
    }
    
    REDUCEERROR(loss)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedHingeError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{

    // Calculate initial offsets
    pUnit                      += blockIdx.x * stride;
    uint64_t dpos               = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
    pData                      += dpos * stride;

    // Calculate loss
    uint32_t pos                = threadIdx.x;
    NNFloat loss                = (NNFloat)0.0;
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
    
    while (pos < stride)
    {
        NNFloat t               = (NNFloat)pData[pos] * (NNFloat)(1.0 / 128.0);
        NNFloat y               = pUnit[pos];
        loss                   += w * max((NNFloat)0.0, (NNFloat)1.0 - t * y);
        pos                    += blockDim.x;    
    }
    
    REDUCEERROR(loss)
}

template<typename T> NNFloat kCalculateIndexedHingeError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    unsigned long threads = max(32, min(stride, 128));
    kCalculateIndexedHingeError_kernel<<<batch, threads>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedHingeError_kernel");    
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE); 
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (-t * log(max(MIN_ERROR, a)));  
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateMultinomialCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}


template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        error                   = w * (-t * log(max(MIN_ERROR, a)));  
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        error                   = w * (-t * log(max(MIN_ERROR, a)));     
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedMultinomialCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedMultinomialCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedMultinomialCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        if (((t == (T)1.0) && (a < cData._SMCE_oneTarget)) || 
            ((t == (T)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        if (((t == (NNFloat)1.0) && (a < cData._SMCE_oneTarget)) || ((t == (NNFloat)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));  
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        if (((t == (NNFloat)1.0) && (a < cData._SMCE_oneTarget)) || ((t == (NNFloat)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));  
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateScaledMarginalCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        if (((t == (T)1.0) && (a < cData._SMCE_oneTarget)) || 
            ((t == (T)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ( (NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));     
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        if (((t == (NNFloat)1.0) && (a < cData._SMCE_oneTarget)) || ((t == (NNFloat)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));  
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        if (((t == (NNFloat)1.0) && (a < cData._SMCE_oneTarget)) || ((t == (NNFloat)0.0) && (a > cData._SMCE_zeroTarget)))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)) - ((NNFloat)1.0 - t) * cData._SMCE_zeroScale * log(max(MIN_ERROR, (NNFloat)1.0 - a)));  
        //printf("%d %llu %f %f %f\n", position, pos, a, t, error);
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedScaledMarginalCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        if ((t != (T)0.0) && (a < cData._SMCE_oneTarget)) 
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        if ((t != (NNFloat)0.0) && (a < cData._SMCE_oneTarget)) 
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));  
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        if ((t != (NNFloat)0.0) && (a < cData._SMCE_oneTarget))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));  
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateMultinomialScaledMarginalCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData, pDataWeight);
    LAUNCHERROR("kCalculateMultinomialScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        T t                     = pData[dOffset + pos];
        if ((t != (T)0.0) && (a < cData._SMCE_oneTarget)) 
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 128.0);
        if ((t != (NNFloat)0.0) && (a < cData._SMCE_oneTarget)) 
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));  
    }

    REDUCEERROR(error)
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedMultinomialScaledMarginalCrossEntropyError_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData, NNFloat* pDataWeight)
{
    uint64_t pos                = (blockIdx.y * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < stride)
    {
        uint64_t uOffset        = blockIdx.x * stride;
        uint64_t dpos           = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];
        uint64_t dOffset        = dpos * stride;
        NNFloat w               = (pDataWeight != NULL) ? pDataWeight[dpos] : (NNFloat)1.0;
        NNFloat a               = pUnit[uOffset + pos];
        NNFloat t               = (NNFloat)pData[dOffset + pos] * (NNFloat)(1.0 / 256.0);
        if ((t != (NNFloat)0.0) && (a < cData._SMCE_oneTarget))
            error               = w * (-t * cData._SMCE_oneScale * log(max(MIN_ERROR, a)));  
    }

    REDUCEERROR(error)
}

template<typename T> NNFloat kCalculateIndexedMultinomialScaledMarginalCrossEntropyError(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData, NNFloat* pDataWeight)
{
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kCalculateIndexedMultinomialScaledMarginalCrossEntropyError_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData, pDataWeight);
    LAUNCHERROR("kCalculateIndexedMultinomialScaledMarginalCrossEntropyError_kernel");
    getGpu()._pbAccumulator->Download();
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);
}

// Instantiates allowable templated functions so we can hide the implementations here
// instead of in the header file because we're mixing CUDA and C++ and that's
// a migraine headache in the making otherwise.
#define EXPLICITLY_INSTANTIATE_KERNELS(T)                                                                                                                                                                  \
template NNFloat kCalculateL1Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                                               \
template NNFloat kCalculateIndexedL1Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                                             \
template NNFloat kCalculateL2Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                                               \
template NNFloat kCalculateIndexedL2Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                                             \
template NNFloat kCalculateL2HingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                                          \
template NNFloat kCalculateIndexedL2HingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                                        \
template NNFloat kCalculateCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                                     \
template NNFloat kCalculateIndexedCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                                   \
template NNFloat kCalculateScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                       \
template NNFloat kCalculateIndexedScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                     \
template NNFloat kCalculateMultinomialCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                          \
template NNFloat kCalculateIndexedMultinomialCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                        \
template NNFloat kCalculateMultinomialScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                            \
template NNFloat kCalculateIndexedMultinomialScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                          \
template NNFloat kCalculateHingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*, NNFloat*);                                                                                                            \
template NNFloat kCalculateIndexedHingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*, NNFloat*);                                                                                          \
template NNFloat kCalculateSparseAnalogL1Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                                                \
template NNFloat kCalculateIndexedSparseAnalogL1Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                              \
template NNFloat kCalculateSparseAnalogL2Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                                                \
template NNFloat kCalculateIndexedSparseAnalogL2Error<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                              \
template NNFloat kCalculateSparseAnalogL2HingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                                           \
template NNFloat kCalculateIndexedSparseAnalogL2HingeError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*, bool);                         \
template NNFloat kCalculateSparseAnalogMultinomialCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*);                                 \
template NNFloat kCalculateIndexedSparseAnalogMultinomialCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*);               \
template NNFloat kCalculateSparseAnalogMultinomialScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*);                   \
template NNFloat kCalculateIndexedSparseAnalogMultinomialScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat* pDataWeight, T*); \
template NNFloat kCalculateSparseDataScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, T*, bool);                                                \
template NNFloat kCalculateIndexedSparseDataScaledMarginalCrossEntropyError<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, T*, bool);                              \
/**/

EXPLICITLY_INSTANTIATE_KERNELS(NNFloat)
EXPLICITLY_INSTANTIATE_KERNELS(double)
EXPLICITLY_INSTANTIATE_KERNELS(unsigned char)
EXPLICITLY_INSTANTIATE_KERNELS(char)
EXPLICITLY_INSTANTIATE_KERNELS(uint32_t)
EXPLICITLY_INSTANTIATE_KERNELS(uint64_t)
EXPLICITLY_INSTANTIATE_KERNELS(int32_t)
EXPLICITLY_INSTANTIATE_KERNELS(int64_t)
